#include "hip/hip_runtime.h"
#define INCLUDE_FROM_CUDA_FILE
#include "PolyEvalGF2nBN.cuh"
#undef INCLUDE_FROM_CUDA_FILE

#ifdef CUDA_SANITY_CHECKS
#include "../utils.hpp"
#endif

#include "CudaUtils.h"

////////////////////////////////////////////////////////////////////////////////
/*
	Some constant memory variables
*/
////////////////////////////////////////////////////////////////////////////////
// Device side
__constant__ sfixn dMaxThreadsPerBlock;
__constant__ sfixn dSharedMemPerBlock;

__global__ void testMontgMult(sfixn* a, sfixn* b, sfixn num_chunks,sfixn* irred_poly, sfixn* mask,sfixn* tmp,sfixn* res) {

	cudaMontgMulBN(a, b, num_chunks, irred_poly, mask, tmp, res);
	//cudaMontgMulBN(NULL, NULL, 0, NULL, NULL, NULL, NULL);
	//cudaBitCheckBN(a, num_chunks);
}

__global__ void testShift(sfixn* x, sfixn num_chunks) {
	cudaBitShiftLeft1BN(x, num_chunks);
}


sfixn *dx, *dCoeffs, *dIrred_poly, *dMask, *dTmp1, *dTmp2, *dTmp_long, *dTmp_Result;

////////////////////////////////////////////////////////////////////////////////
/*
	Host Functions
*/
////////////////////////////////////////////////////////////////////////////////

CUDA_CALLABLE_MEMBER GF2nPolyBN::GF2nPolyBN(
	sfixn* coeffs, 
	sfixn* x,
	sfixn num_x,
	sfixn size_field,
	sfixn deg_poly, 
	sfixn* irred_poly,
	sfixn* mask ) {

	m_num_x = num_x;
	m_size_field = size_field;

	// How many coefficients does our polynomial have
	m_num_coeffs = deg_poly + 1;

	// Calculate the number of chunks of the data elements
	m_num_chunks = getNumberChunks(m_size_field + 1);

	// Calcualte the amount of leaves needed for used binary tree method
	m_width_binary_tree = getCeilToPotOf2n(m_num_coeffs);

	// How many bytes do we need for one field element
	m_bytes_for_chunks = m_num_chunks * SIZE_CHUNK / SIZE_BYTE;

#ifdef CUDA_SANITY_CHECKS
	std::cout << "Creating Result file..." << std::endl;

	remove("rsh_test_results");

	m_result_file = new std::ofstream();
	m_result_file->open("rsh_test_results");

	// write input parameters
	(*m_result_file) << "field_size=";
	printbinToFile(&m_size_field, 1, 1, m_result_file);
	(*m_result_file) << "\n";
	(*m_result_file) << "num_coeffs=";
	printbinToFile(&m_num_coeffs, 1, 1, m_result_file);
	(*m_result_file) << "\n";
	(*m_result_file) << "coeffs=";
	printbinToFile(coeffs, m_num_chunks, m_num_coeffs, m_result_file);
	(*m_result_file) << "\n";
	(*m_result_file) << "x=";
	printbinToFile(x, m_num_chunks, 1, m_result_file);
	(*m_result_file) << "\n";
	(*m_result_file) << "irred_poly=";
	printbinToFile(irred_poly, m_num_chunks, 1, m_result_file);
	(*m_result_file) << "\n";
	(*m_result_file) << "mask=";
	printbinToFile(mask, m_num_chunks, 1, m_result_file);
	(*m_result_file) << "\n";
#endif

	loadPoroperties();

	// Allocate global memory
	CudaSafeCall(hipMalloc((sfixn**)&m_dx, m_num_x * m_bytes_for_chunks));
	CudaSafeCall(hipMalloc((sfixn**)&m_dCoeffs, m_width_binary_tree * m_bytes_for_chunks));
	CudaSafeCall(hipMalloc((sfixn**)&m_dIrred_poly, m_bytes_for_chunks));
	CudaSafeCall(hipMalloc((sfixn**)&m_dMask, m_bytes_for_chunks));
	CudaSafeCall(hipMalloc((sfixn**)&m_dTmp1, m_width_binary_tree / 2 * m_bytes_for_chunks));
	CudaSafeCall(hipMalloc((sfixn**)&m_dTmp2, m_width_binary_tree / 2 * m_bytes_for_chunks));
	CudaSafeCall(hipMalloc((sfixn**)&m_dTmp_long, m_width_binary_tree * m_bytes_for_chunks));
	CudaSafeCall(hipMalloc((sfixn**)&m_dTmp_Result, m_width_binary_tree * m_bytes_for_chunks));

	sfixn hCoeffs[m_width_binary_tree * m_bytes_for_chunks];
	padWithZeros(coeffs, m_num_coeffs, m_num_chunks, hCoeffs, m_width_binary_tree);

	// Copy host data to device
	CudaSafeCall(hipMemcpy(m_dx, x, m_num_x * m_bytes_for_chunks, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(m_dCoeffs, hCoeffs, m_width_binary_tree * m_bytes_for_chunks, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(m_dIrred_poly, irred_poly, m_bytes_for_chunks, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(m_dMask, mask, m_bytes_for_chunks, hipMemcpyHostToDevice));

	m_hTmp_Result = new sfixn[m_width_binary_tree * m_bytes_for_chunks];

}

CUDA_CALLABLE_MEMBER GF2nPolyBN::~GF2nPolyBN() {

	// Free all allocated device memory
	CudaSafeCall(hipFree(m_dx));
	CudaSafeCall(hipFree(m_dCoeffs));
	CudaSafeCall(hipFree(m_dIrred_poly));
	CudaSafeCall(hipFree(m_dMask));
	CudaSafeCall(hipFree(m_dTmp1));
	CudaSafeCall(hipFree(m_dTmp2));
	CudaSafeCall(hipFree(m_dTmp_long));
	CudaSafeCall(hipFree(m_dTmp_Result));

	delete m_hTmp_Result;

#ifdef CUDA_SANITY_CHECKS
	std::cout << "finished" << std::endl;
	m_result_file->close();
	delete m_result_file;
#endif

}

////////////////////////////////////////////////////////////////////////////////
//
//	A polynomial evaluation over GF(2n) with BN Coefficients
//
//	Evaluates a polynomial with BN coefficients in GF(2n)
// 	at point x. Note that the size of the field elements (coeffs, x, result)
//	must be big enough to store an additional carry bit (size = size_field + 1)!
//
//	Input:
//		coeffs 			- the coefficients of the polynomial		-> size = (k + 1) * (n + 1)
//		x 				- point where the polynomial is evaluated	-> size = n + 1
//		size_field  	- number of bits of the field elements		-> n
//		deg_poly 		- degree of the evaluated polynomial 		-> k
//		irred_poly 		- the irreducible polynomial of the GF(2n) 	-> size = n + 1
// 		mask 			- mask of the field 						-> size = n + 1
//		result			- the result 								-> size = n + 1
//
////////////////////////////////////////////////////////////////////////////////
CUDA_CALLABLE_MEMBER void GF2nPolyBN::evaluate(sfixn i) {

	sfixn num_threads, num_blocks;

	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	num_threads = min(m_num_chunks*m_width_binary_tree, m_hMaxThreadsPerBlock);
	num_blocks = ceil((double)m_num_chunks*m_width_binary_tree/m_hMaxThreadsPerBlock);
	printf("Starting expand step...\n");
	cudaExpandVecBNKernel<<<num_blocks, num_threads>>>(&m_dx[i * m_num_chunks], m_num_chunks, m_dTmp_Result, m_num_chunks*m_width_binary_tree);
	hipDeviceSynchronize();
#ifdef CUDA_SANITY_CHECKS
	CudaSafeCall(hipMemcpy(m_hTmp_Result, m_dTmp_Result, m_width_binary_tree * m_bytes_for_chunks, hipMemcpyDeviceToHost));
	(*m_result_file) << "resultExpandStep=";
	printbinToFile(m_hTmp_Result, m_num_chunks, m_width_binary_tree, m_result_file);
	(*m_result_file) << "\n";
#endif

	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	// Create all exponentiation from x^0 to x^deg_poly and store it in res 
	num_threads = min(m_width_binary_tree/2, m_hMaxThreadsPerBlock);
	num_blocks = ceil((double)m_width_binary_tree/2/m_hMaxThreadsPerBlock);
	// Calculate reduce step
	printf("Starting reduce step...\n");
	cudaPrefProdReduce<<<num_blocks, num_threads>>>(m_num_chunks, m_dIrred_poly, m_dMask, m_width_binary_tree, m_dTmp1, m_dTmp_Result);
	hipDeviceSynchronize();
#ifdef CUDA_SANITY_CHECKS
	CudaSafeCall(hipMemcpy(m_hTmp_Result, m_dTmp_Result, m_width_binary_tree * m_bytes_for_chunks, hipMemcpyDeviceToHost));
	(*m_result_file) << "resultReduceStep=";
	printbinToFile(m_hTmp_Result, m_num_chunks, m_width_binary_tree, m_result_file);
	(*m_result_file) << "\n";
#endif

	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	// calculate down sweep step
	printf("Starting down swep step...\n");
	cudaPrefProdDownSweep<<<num_blocks, num_threads>>>(m_num_chunks, m_dIrred_poly, m_dMask, m_width_binary_tree, m_dTmp1, m_dTmp2, m_dTmp_Result);
	hipDeviceSynchronize();
#ifdef CUDA_SANITY_CHECKS
	CudaSafeCall(hipMemcpy(m_hTmp_Result, m_dTmp_Result, m_width_binary_tree * m_bytes_for_chunks, hipMemcpyDeviceToHost));
	(*m_result_file) << "resultDownSweepStep=";
	printbinToFile(m_hTmp_Result, m_num_chunks, m_width_binary_tree, m_result_file);
	(*m_result_file) << "\n";
#endif

	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	// Multiply each coefficient with its related exponentiation of x 
	// (coeff[0]*x^0, ..., coeff[deg_poly]*x^deg_poly) and store it in res
	num_threads = min(m_width_binary_tree, m_hMaxThreadsPerBlock);
	num_blocks = ceil((double)m_width_binary_tree/m_hMaxThreadsPerBlock);

	printf("Starting prod step...\n");
	cudaMontgMulBNKernel<<<num_blocks, num_threads>>>(m_dCoeffs, m_dTmp_Result, m_width_binary_tree, m_num_chunks, m_dIrred_poly, m_dMask, m_dTmp_long, m_dTmp_Result);
	hipDeviceSynchronize();
#ifdef CUDA_SANITY_CHECKS
	CudaSafeCall(hipMemcpy(m_hTmp_Result, m_dTmp_Result, m_width_binary_tree * m_bytes_for_chunks, hipMemcpyDeviceToHost));
	(*m_result_file) << "resultProdStep=";
	printbinToFile(m_hTmp_Result, m_num_chunks, m_width_binary_tree, m_result_file);
	(*m_result_file) << "\n";
#endif

	// Add all summands of the polynom up to the result
	num_threads = min(m_width_binary_tree/2, m_hMaxThreadsPerBlock);
	num_blocks = ceil((double)m_width_binary_tree/2/m_hMaxThreadsPerBlock);

	printf("Starting sum step...\n");
	cudaBitSumBNKernel<<<num_blocks, num_threads>>>(m_dTmp_Result, m_num_chunks, m_width_binary_tree);
	hipDeviceSynchronize();

#ifdef CUDA_SANITY_CHECKS
	CudaSafeCall(hipMemcpy(m_hTmp_Result, m_dTmp_Result, m_width_binary_tree * m_bytes_for_chunks, hipMemcpyDeviceToHost));
	(*m_result_file) << "resultSumStep=";
	printbinToFile(m_hTmp_Result, m_num_chunks, 1, m_result_file);
	(*m_result_file) << "\n";
#endif	

}

CUDA_CALLABLE_MEMBER void GF2nPolyBN::getResults(sfixn *result) {

	// Read the result from device
	CudaSafeCall(hipMemcpy(m_hTmp_Result, m_dTmp_Result, m_width_binary_tree * m_bytes_for_chunks, hipMemcpyDeviceToHost));

	// Create result
	for( sfixn i=0; i<m_num_chunks; ++i ) {
		result[i] = m_hTmp_Result[i];
	}

}

////////////////////////////////////////////////////////////////////////////////
//
//	Load some cuda properties to constant memory
//
////////////////////////////////////////////////////////////////////////////////
CUDA_CALLABLE_MEMBER void GF2nPolyBN::loadPoroperties() {

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dMaxThreadsPerBlock), (const char *)&deviceProp.maxThreadsPerBlock, sizeof(sfixn), 0, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dSharedMemPerBlock), &deviceProp.sharedMemPerBlock, sizeof(sfixn)));

	m_hMaxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
	m_hSharedMemPerBlock = deviceProp.sharedMemPerBlock;
}

////////////////////////////////////////////////////////////////////////////////
//
//	Adds leading zero blocks to the array so that its new size matches size_new
//
////////////////////////////////////////////////////////////////////////////////
__host__ void padWithZeros( 
	sfixn* data_old, 
	sfixn size_old, 
	sfixn num_chunks, 
	sfixn* data_new, 
	sfixn size_new ) {

	for( sfixn i=size_new*num_chunks-1; i>=0; --i ) {
		if( i >= size_new - size_old ) {
			data_new[i] = data_old[i - (size_new - size_old)];
		} else {
			data_new[i] = 0;
		}
	}
}

////////////////////////////////////////////////////////////////////////////////
//
//	Returns the next higher pot of 2 of the passed value
//
////////////////////////////////////////////////////////////////////////////////
__host__ sfixn getCeilToPotOf2n( sfixn value ) {
	
	sfixn res = 0;
	
	if( value > 0 ) {
		res = 1;
		while(res < value) {
			res<<=1;
		}
	}

	return res;
}

__host__ sfixn getNumberBlocksForSharedMem( sfixn sharedMemSize ) {

	return 0;
}

////////////////////////////////////////////////////////////////////////////////
/*
	Kernel Functions
*/
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//
//	Callculates all exponentiation of x to the power of n
//
//	Input:
//		x 				- point where the polynomial is evaluated
//		length_x	 	- number of bits of x
//		irred_poly 		- the irreducible polynomial of the GF(2n)
//		y				- the result
//
////////////////////////////////////////////////////////////////////////////////
__global__ void cudaPrefProdReduce(
	sfixn num_chunks,
	sfixn* irred_poly,
	sfixn* mask, 
	sfixn length_exp_tree,
	sfixn* tmp,
	sfixn* res) {

    sfixn thid = (blockIdx.x * blockDim.x) + threadIdx.x;

    if( thid < length_exp_tree/2 ) {

		sfixn *local_tmp = &tmp[thid * num_chunks];			

	    sfixn offset = 1;

	    int i = 0;

	 	for( sfixn d=length_exp_tree>>1; d>0; d>>=1 ) { //build sum in place up the tree 
	 		__syncthreads();

	 		if( thid < d ) {
	 			sfixn ai = (offset * (2*thid+1) - offset) * num_chunks;
	 			sfixn bi = (offset * (2*thid+2) - offset) * num_chunks;

				cudaMontgMulBN(
					&res[ai], 
					&res[bi],
					num_chunks,
					irred_poly,
					mask,
					local_tmp,
					&res[ai]);

				__syncthreads();
	 		}
	 		__syncthreads();

	 		offset <<= 1;

	 		++i;
		}
	}
}

////////////////////////////////////////////////////////////////////////////////
//
//	Callculates all exponentiation of x to the power of n
//
//	Input:
//		x 				- point where the polynomial is evaluated
//		length_x	 	- number of bits of x
//		irred_poly 		- the irreducible polynomial of the GF(2n)
//		y				- the result
//
////////////////////////////////////////////////////////////////////////////////
__global__ void cudaPrefProdDownSweep(
	sfixn num_chunks,
	sfixn* irred_poly,
	sfixn* mask, 
	sfixn length_exp_tree,
	sfixn* tmp1,
	sfixn* tmp2,
	sfixn* res) {

    sfixn thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	sfixn num_threads, num_blocks;

    if( thid < length_exp_tree/2 ) {

		sfixn *local_tmp1 = &tmp1[thid * num_chunks];
		sfixn *local_tmp2 = &tmp2[thid * num_chunks];

		num_threads = min(num_chunks, dMaxThreadsPerBlock);
		num_blocks = ceil((double)num_chunks/dMaxThreadsPerBlock);
		
		if( thid == 0 ) {
			cudaSet0Kernel<<<num_blocks, num_threads>>>(&local_tmp1[0], num_chunks);
			cudaSet1Kernel<<<num_blocks, num_threads>>>(&res[0], num_chunks);
		}

		sfixn offset = length_exp_tree;

		sfixn i = 0;

		for( sfixn d=1; d<length_exp_tree; d*=2 ) { //traverse down tree & build scan 

		    offset >>= 1;
		    __syncthreads();

			if( thid < d ) {
				sfixn ai = (offset * (2*thid+1) - offset) * num_chunks;
		 		sfixn bi = (offset * (2*thid+2) - offset) * num_chunks;

		 		__syncthreads();

				cudaCopyBNKernel<<<num_blocks, num_threads>>>(&res[bi], num_chunks, local_tmp1, num_chunks);
				__syncthreads();
				if( thid == 0 ) {
					//printf("local_tmp1 in iteration i: "); cudaPrintbincharpad(local_tmp1, num_chunks);
				}
				cudaCopyBNKernel<<<num_blocks, num_threads>>>(&res[ai], num_chunks, &res[bi], num_chunks);
				__syncthreads();

				cudaMontgMulBN(
					&res[ai], 
					local_tmp1,
					num_chunks,
					irred_poly,
					mask,
					local_tmp2,
					&res[ai]);
				__syncthreads();

				++i;
		    }
		}

		__syncthreads();
	}
}

////////////////////////////////////////////////////////////////////////////////
//
//	Shifts a n bits to the right
//
////////////////////////////////////////////////////////////////////////////////
__global__ void cudaBitShiftRightBNKernel(sfixn* a, sfixn length_a, sfixn n, sfixn* c) {

	sfixn thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(thid < getNumberChunks(length_a)) {

		sfixn min_range = thid * SIZE_CHUNK; //<- Block size
		//sfixn max_range = min_range + SIZE_CHUNK - 1;

		sfixn min_src_range = min_range - n;
		sfixn max_src_range = min_src_range + SIZE_CHUNK - 1;

		if(max_src_range < 0) {
			c[thid] = 0;
		} else {
			sfixn digits_block_b_shift = (max_src_range + 1) % SIZE_CHUNK;
			if(min_src_range < 0) {
				sfixn mask_b = pow((double)2, (double)SIZE_CHUNK - 1) - pow((double)2, (double)(SIZE_CHUNK-(max_src_range%SIZE_CHUNK)) - 1);
				sfixn src_idx_b = max_src_range/SIZE_CHUNK;
				c[thid] = ((a[src_idx_b]&mask_b) >> (SIZE_CHUNK-(max_src_range%SIZE_CHUNK)));
			} else if(digits_block_b_shift == 0) {
				c[thid] = a[min_src_range/SIZE_CHUNK];
			} else {
				sfixn mask_a = pow((double)2, (double)(SIZE_CHUNK-(min_src_range%SIZE_CHUNK))) - 1;
				sfixn mask_b = (pow((double)2, (double)SIZE_CHUNK) - 1) - (pow((double)2, (double)(SIZE_CHUNK-(max_src_range%SIZE_CHUNK))) - 1);

				sfixn src_idx_a = min_src_range/SIZE_CHUNK;
				sfixn src_idx_b = max_src_range/SIZE_CHUNK;

				c[thid] = ((a[src_idx_a]&mask_a) << (min_src_range%SIZE_CHUNK)) | 
					((a[src_idx_b]&mask_b) >> (SIZE_CHUNK-(max_src_range%SIZE_CHUNK)));

			}
		}
	}
}

////////////////////////////////////////////////////////////////////////////////
//
//	Shifts a n bits to the left
//
////////////////////////////////////////////////////////////////////////////////
__global__ void cudaBitShiftLeftBNKernel(sfixn* a, sfixn num_chunks, sfixn n, sfixn* c) {

	sfixn thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( thid < num_chunks ) {

		sfixn min_range = thid * SIZE_CHUNK; //<- Block size
		//sfixn max_range = min_range + SIZE_CHUNK - 1;

		sfixn min_src_range = min_range + n;
		sfixn max_src_range = min_src_range + SIZE_CHUNK - 1;

		if( min_src_range >= num_chunks * SIZE_CHUNK ) {
			c[thid] = 0;
		} else {
			bool shift_whole_block = ((max_src_range + 1) % SIZE_CHUNK == 0);
			if( max_src_range >= num_chunks * SIZE_CHUNK ) {
				sfixn mask_b = pow((double)2, (double)(SIZE_CHUNK-(min_src_range % SIZE_CHUNK))) - 1;
				sfixn src_idx_b = min_src_range/SIZE_CHUNK;
				c[thid] = ((a[src_idx_b]&mask_b) << (min_src_range%SIZE_CHUNK));
			} else if( shift_whole_block ) {
				c[thid] = a[min_src_range/SIZE_CHUNK];
			} else {
				sfixn mask_a = pow((double)2, (double)(SIZE_CHUNK-(min_src_range%SIZE_CHUNK))) - 1;
				sfixn mask_b = (pow((double)2, (double)SIZE_CHUNK) - 1) - (pow((double)2, (double)(SIZE_CHUNK-((max_src_range+1)%SIZE_CHUNK))) - 1);

				sfixn src_idx_a = min_src_range/SIZE_CHUNK;
				sfixn src_idx_b = max_src_range/SIZE_CHUNK;

				c[thid] = ((a[src_idx_a]&mask_a) << (min_src_range%SIZE_CHUNK)) | 
					((a[src_idx_b]&mask_b) >> (SIZE_CHUNK-((max_src_range+1)%SIZE_CHUNK)));

			}
		}
	}
}

////////////////////////////////////////////////////////////////////////////////
//
//	Copies a to b
//
////////////////////////////////////////////////////////////////////////////////
__global__ void cudaCopyBNKernel( sfixn* a, sfixn num_chunks_a, sfixn* b, sfixn num_chunks_b ) {

	sfixn thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( thid < num_chunks_b ) {
		if( thid < num_chunks_a )
			b[thid] = a[thid];
		else
			b[thid] = 0;
	}
}

////////////////////////////////////////////////////////////////////////////////
//
//	A binary montgomery multiplication Kernel over big numbers
//
////////////////////////////////////////////////////////////////////////////////
__global__ void cudaMontgMulBNKernel(
	sfixn* values,
	sfixn* x_preCalc,
	sfixn num_values,
	sfixn num_chunks, 
	sfixn* irred_poly,
	sfixn* mask,
	sfixn* tmp,
	sfixn* res) {

	sfixn thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	// if(thid == 0) {
	// 	printf("values: \n"); 
	// 	for(int i=0; i<num_values; ++i) {
	// 		printf("values[%i]: ", i); cudaPrintbincharpad(&values[i], num_chunks);
	// 	}
	// 	printf("x_preCalc: \n"); 
	// 	for(int i=0; i<num_values; ++i) {
	// 		printf("x_preCalc[%i]: ", i); cudaPrintbincharpad(&x_preCalc[i], num_chunks);
	// 	}
	//}

	cudaMontgMulBN(
		&values[thid*num_chunks], 
		&x_preCalc[thid*num_chunks],
		num_chunks,
		irred_poly,
		mask,
		&tmp[thid*num_chunks],
		&res[thid*num_chunks]);
}

////////////////////////////////////////////////////////////////////////////////
//
//	Sums up all passed values
//
////////////////////////////////////////////////////////////////////////////////
__global__ void cudaBitSumBNKernel(
	sfixn* values, 
	sfixn num_chunks,
	sfixn n ) {

	sfixn thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	int i = 0;

	if( thid < n/2 ) {
		sfixn offset = 1;

		for( sfixn d=n>>1; d>0; d>>=1 ) {
	 		__syncthreads();

	 		if( thid < d ) {
	 			sfixn ai = (offset * (2*thid+1) - offset) * num_chunks;
	 			sfixn bi = (offset * (2*thid+2) - offset) * num_chunks;

	 			// Calculate values_bi = values_ai + values_bi
				sfixn num_threads = min(num_chunks, dMaxThreadsPerBlock);
				sfixn num_blocks = ceil((double)num_chunks/dMaxThreadsPerBlock);
				cudaBitAddBNKernel<<<num_blocks, num_threads>>>(&values[ai], &values[bi], num_chunks);

				__syncthreads();
	 		}
	 		++i;
	 		offset <<= 1;
		}
	}
}

////////////////////////////////////////////////////////////////////////////////
//
//	Creates a vector that contains n instances of the value
//
////////////////////////////////////////////////////////////////////////////////
__global__ void cudaExpandVecBNKernel(
	sfixn* value,
	sfixn blocks_per_value,
	sfixn* value_vec,
	sfixn blocks_value_vec
	) {

	sfixn thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( thid < blocks_value_vec ) {
		value_vec[thid] = value[thid % blocks_per_value];
	}
}

////////////////////////////////////////////////////////////////////////////////
//
//	Sets x to 0
//
////////////////////////////////////////////////////////////////////////////////
__global__ void cudaSet0Kernel( sfixn* x, sfixn length ) {

	sfixn thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( thid < length )
		x[thid] = 0;
}

////////////////////////////////////////////////////////////////////////////////
//
//	Sets x to 1
//
////////////////////////////////////////////////////////////////////////////////
__global__ void cudaSet1Kernel( sfixn* x, sfixn length ) {

	sfixn thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( thid < length )
		if( thid == length-1 )
			x[thid] = 1;
		else
			x[thid] = 0;
}

////////////////////////////////////////////////////////////////////////////////
//
//	Adds a and b and stores the result in a
//
////////////////////////////////////////////////////////////////////////////////
__global__ void cudaBitAddBNKernel(sfixn* a, sfixn* b, sfixn num_chunks) {

	sfixn thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( thid < num_chunks )
		a[thid] ^= b[thid];
}

////////////////////////////////////////////////////////////////////////////////
//
//	Adds a and b and stores the result in c
//
////////////////////////////////////////////////////////////////////////////////
__global__ void cudaBitAndBNKernel(sfixn* a, sfixn* b, sfixn* c, sfixn num_chunks) {

	sfixn thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( thid < num_chunks )
		c[thid] = a[thid] & b[thid];
}

////////////////////////////////////////////////////////////////////////////////
/*
	Device Functions
*/
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//
//	A binary montgomery multiplication over big numbers
//
////////////////////////////////////////////////////////////////////////////////
__device__ void cudaMontgMulBN(
	sfixn* a, 
	sfixn* b, 
	sfixn num_chunks,
	sfixn* irred_poly, 
	sfixn* mask,
	sfixn* tmp,
	sfixn* res) {

	sfixn num_threads = min(num_chunks, dMaxThreadsPerBlock);
	sfixn num_blocks = ceil((double)num_chunks/dMaxThreadsPerBlock);

	cudaSet0Kernel<<<num_blocks, num_threads>>>(tmp, num_chunks);

	int iteration = 0;

	// if( threadIdx.x == 100 ) {	
	// 	printf("thread 1 started...\n");
	// 	printf("a: "); cudaPrintbincharpad(a, num_chunks);
	// 	printf("b: "); cudaPrintbincharpad(b, num_chunks);
	// 	printf("res: "); cudaPrintbincharpad(res, num_chunks);
	// 	printf("tmp: "); cudaPrintbincharpad(tmp, num_chunks);
	// 	printf("mask: "); cudaPrintbincharpad(mask, num_chunks);
	// 	printf("irred_poly: "); cudaPrintbincharpad(irred_poly, num_chunks);
	// 	printf("dMaxThreadsPerBlock: %i\n", dMaxThreadsPerBlock);
	// 	printf("num_chunks: %i\n", num_chunks); 
	// 	printf("num blocks: %i\n", num_blocks);
	// 	printf("num threads: %i\n", num_threads);
	// }

	bool hit = 0;

	for( sfixn i=num_chunks-1; i>=0; --i ) {

		//__syncthreads();

		for( sfixn j=0; j<SIZE_CHUNK; ++j ) {
		
			//printf("Tmp in iteration %i: ", j); cudaPrintbincharpad(tmp, num_chunks);
			//printf("a in iteration %i: ", j); cudaPrintbincharpad(a, num_chunks);

			//__syncthreads();
			// if(i == num_chunks-1 && j<10) {
			// 	//printf("b[%i]: ", i); cudaPrintbincharpad(&b[i], 1);
			// 	printf("bit: %i\n", j);
			// }
			if( isbitset(b[i], j) ) {
				hit = true;
				//f(i == num_chunks-1)
					//printf("hit\n");
				cudaBitAddBNKernel<<<num_blocks, num_threads>>>(tmp, a, num_chunks);
			} else {
				hit = false;
				//if(i == num_chunks-1 )
					//printf("no hit\n");
			}

			//__syncthreads();
			
			// TODO: There seems to be a bug here if the printf is removed.
			// 		 In this case tmp is shifted after the addition for an unknown reason
			//printf("");
			// if(i == num_chunks-1 && j<10 && hit) {
			// 	printf("a before: ", j); cudaPrintbincharpad(a, num_chunks);
			// }

			cudaBitShiftLeft1BN(a, num_chunks);
			// if(i == num_chunks-1 && j<10 && hit) {
			// 	printf("a after shift: "); cudaPrintbincharpad(a, num_chunks);
			// }
			//__syncthreads();
			cudaReducePolyBN(a, num_chunks, irred_poly, mask);
			//__syncthreads();
						
			//printf("result 2 in iteration %i: ", j); cudaPrintbincharpad(tmp, num_chunks);

			//__syncthreads();
			// if(i == num_chunks-1 && j<10) {
			// 	printf("result: ", j); cudaPrintbincharpad(tmp, num_chunks);
			// 	if(hit) {
			// 		printf("a: ", j); cudaPrintbincharpad(a, num_chunks);
			// 	}	
			// }
		}
	}

	cudaCopyBNKernel<<<num_blocks, num_threads>>>(tmp, num_chunks, res, num_chunks);
}

////////////////////////////////////////////////////////////////////////////////
//
//	Reduce the extended field polynomial
//
////////////////////////////////////////////////////////////////////////////////
__device__ void cudaReducePolyBN(
	sfixn* value,
	sfixn num_chunks,
	sfixn* irred_poly,
	sfixn* mask
	) {

	if( (value[0] & mask[0]) ) {
		sfixn num_threads = min(num_chunks, dMaxThreadsPerBlock);
		sfixn num_blocks = ceil((double)num_chunks/dMaxThreadsPerBlock);
		if(threadIdx.x == 0) {
			//printf("hit\n");
		} 
		cudaBitAddBNKernel<<<num_blocks, num_threads>>>(value, irred_poly, num_chunks);
	} else {
		if(threadIdx.x == 0) {
			//printf("no hit\n");
		}
	}
}

////////////////////////////////////////////////////////////////////////////////
//
//	Checks if a is not 0
//
////////////////////////////////////////////////////////////////////////////////
__device__ bool cudaBitCheckBN( sfixn* x, sfixn num_chunks ) {

	sfixn result = false;

	for( sfixn i=0; i<num_chunks; i++ ) {
		if( x[i] > 0 ) {
			result = true;
			break;
		}
	}
	return result;
}

////////////////////////////////////////////////////////////////////////////////
//
//	Shift a 1 Bit to the left
//
////////////////////////////////////////////////////////////////////////////////
__device__ void cudaBitShiftLeft1BN( sfixn* a, sfixn num_chunks ) {
	longfixnum tmp = 0;
	sfixn carry = 0;

	longfixnum lmask = pow((double)2, (double)SIZE_CHUNK) - 1;
	longfixnum umask = pow((double)2, (double)SIZE_CHUNK);

	for( sfixn i = num_chunks - 1; i >= 0; --i ) {
		tmp = 0;
		tmp = a[i];
		tmp <<= 1;
		a[i] = (tmp&lmask) | carry;
		carry = (tmp&umask) >> SIZE_CHUNK;
	}
}

////////////////////////////////////////////////////////////////////////////////
//
//	Calculates the number of chunks for the diggit length
//
////////////////////////////////////////////////////////////////////////////////
__host__ __device__ sfixn getNumberChunks( sfixn length ) {

	if( !length ) {
		return 0;
	} else {
		return (length-1)/SIZE_CHUNK+1;
	}
} 

////////////////////////////////////////////////////////////////////////////////
//
//	Checks if the bit at pos bitnum is set (idx from right to left)
//
////////////////////////////////////////////////////////////////////////////////
__host__ __device__ sfixn isbitset( sfixn val, sfixn bitnum ) {
	return (val & (1 << bitnum)) != 0;
}

__global__ void cudaPrintbincharpadKernel(sfixn* ca, unsigned int n)
{
	for(int j=0; j<n; j++) {
		sfixn c = ca[j];
	    for (int i = sizeof(sfixn)*8-1; i >= 0; --i)
	    {
	        if(c & (1 << i)) 
			printf("%c", '1');
		else
			printf("%c", '0');
	    }
	    printf("%c", ' ');
	}
	printf("\n");
}

__device__ void cudaPrintbincharpad(sfixn* ca, unsigned int n)
{
	for(int j=0; j<n; j++) {
		sfixn c = ca[j];
	    for (int i = sizeof(sfixn)*8-1; i >= 0; --i)
	    {
	        if(c & (1 << i)) 
			printf("%c", '1');
		else
			printf("%c", '0');
	    }
	    printf("%c", ' ');
	}
	printf("\n");
}
